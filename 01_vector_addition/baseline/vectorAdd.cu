#include <algorithm>
#include <iostream>
#include <vector>


#include <hip/hip_runtime.h>



__global__ void vectorAdd(int* a, int* b, int* c, int n) {

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(tid < n) {
        c[tid] = a[tid] + b[tid];
    }

}


int main() {

   constexpr int N = 100;
    constexpr size_t bytes = sizeof(int) * N;

    std::vector<int> a(N);
    std::vector<int> b(N);
    std::vector<int> c(N);

    // Populate vectors a and b using std::for_each
    int x=0;

    
    std::for_each(a.begin(), a.end(), [&x](int& val) { val = x++; });
    x=0;
    std::for_each(b.begin(), b.end(), [&x](int& val) { val = x++; });

    int* d_a, *d_b, *d_c;
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    // Corrected cudaMemcpy calls
    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    int NUM_THREADS = 1024;
    int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

    vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

    hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Print the output array
    std::cout << "Output array:\n";
    for (int i = 0; i < N; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << "\nCOMPLETED SUCCESSFULLY\n";

    return 0;
}